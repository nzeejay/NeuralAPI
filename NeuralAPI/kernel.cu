#include "hip/hip_runtime.h"
﻿//Kernel code:
extern "C"
{
	__device__ int getIndex(int x, int y, int z, int X, int Y) {
		return (x + (y * Y)) + (z * X * Y);
	}

	__device__ float sigmoid(float f) {
		return 1 / (1 + __expf(-f));
	}

	__device__ float sigmoidDer(float f) {
		return f * (1 - f);
	}

	__device__ float relu(float f) {

		return fmaxf(0, f);
	}

	__device__ float reluDer(float f) {
		float ret = 1;

		if (f <= 0)
			ret = 0;

		return ret;
	}

	//activations
	__global__ void Activate(float* data, float* bias, int activationID) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		switch (activationID)
		{
		case(0):
			data[ID] = sigmoid(data[ID] + bias[ID]);
			break;

		case(1):
			data[ID] = relu(data[ID] + bias[ID]);
			break;
		default:
			break;
		}
	}

	__device__ float activateDer(float f, int activationID) {
		switch (activationID)
		{
		case(0):
			return sigmoidDer(f);
			break;
		case(1):
			return reluDer(f);
			break;
		}
		return 1.f;
	}

	__global__ float SoftmaxSigma(float *data, float ret) {
		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);
		
		atomicAdd(&ret, data[thisLayerID]);
	}

	__global__ void SoftmaxFinal(float *data, float val) {
		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);
		
		data[thisLayerID] /= val; 
	}

	// Device code
	__global__ void Forward(float* data, 
							float* weights, 
						    float* prev) {

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * (blockDim.x * blockDim.y * blockDim.z) + prevLayerID;

		float val = prev[prevLayerID] * weights[weightID];
	
		atomicAdd(&data[thisLayerID], val);
	}

	__global__ void Backprop(float* data, 
							 float* weights, 
							 float* bias, 
							 float* prev, 
							 float* error, 
							 float* prevError, 
							 float* vel, 
							 float step, 
							 float mu, 
							 int type) {
		int blockSize = (blockDim.x * blockDim.y * blockDim.z);

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * blockSize + prevLayerID;

		float gradient = (activateDer(prev[prevLayerID], type) * 2 * error[thisLayerID]);			
		
		bias[thisLayerID] += -step * gradient;

		float velocity = vel[weightID] * mu - step * prev[prevLayerID] * gradient;
		vel[weightID] = velocity;// mu * vel[weightID] + (1 - mu) * powf(gradient, 2);
		weights[weightID] += velocity;// prev[prevLayerID] * gradient / (sqrtf(vel[weightID]) + 0.000001f);

		float prevActGD = weights[weightID] * gradient;
//printf("grad: %f act: %f err: %f prev: %f prevActGD: %f\n", gradient, activateDer(prev[prevLayerID], type), error[thisLayerID], prev[prevLayerID], prevActGD);
		atomicAdd(&prevError[prevLayerID], prevActGD);
	}


	__global__ void Clear(float* data, float* error) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);
		
		data[ID] = 0.f;
		error[ID] = 0.f;
	}
}