
#include <hip/hip_runtime.h>
﻿//Kernel code:
extern "C"
{
	__device__ int getIndex(int x, int y, int z, int X, int Y) {
		return (x + (y * Y)) + (z * X * Y);
	}

	__device__ float sigmoid(float f) {
		return 1 / (1 + __expf(-f));
	}

	__device__ float sigmoidDer(float f) {
		return f * (1 - f);
	}

	__device__ float relu(float f) {
		if (f < 0)
			return 0;

		return f;
	}

	__device__ float reluDer(float f) {
		float ret = -1;

		if (f < 0)
			ret = 0;

		return ret;
	}

	//activations
	__global__ void Activate(float* data, float* bias, int activationID) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		switch (activationID)
		{
		case(0):
			data[ID] = sigmoid(data[ID] + bias[ID]);
			break;

		case(1):
			data[ID] = relu(data[ID] + bias[ID]);
			break;
		
		default:
			break;
		}
	}

	__device__ float activateDer(float f, int activationID) {
		switch (activationID)
		{
		case(0):
			return sigmoidDer(f);
			break;
		case(1):
			return reluDer(f);
			break;
		}

		return 1.f;
	}

	// Device code
	__global__ void Forward(float* data, 
							float* weights, 
						    float* prev) {

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * (blockDim.x * blockDim.y * blockDim.z) + prevLayerID;

		float val = prev[prevLayerID] * weights[weightID];
	
		atomicAdd(&data[thisLayerID], val);
	}

	__global__ void Backprop(float* data, 
							 float* weights, 
							 float* bias, 
							 float* prev, 
							 float* error, 
							 float* prevError, 
							 float* vel, 
							 float step, 
							 float mu, 
							 int type) {
		int blockSize = (blockDim.x * blockDim.y * blockDim.z);

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * blockSize + prevLayerID;

		float gradient = (-step * activateDer(prev[prevLayerID], type) * error[thisLayerID]);
		bias[thisLayerID] += gradient;

		float velocity = vel[weightID] * mu - prev[prevLayerID] * gradient;
		vel[weightID] = velocity;// mu * vel[weightID] + (1 - mu) * powf(gradient, 2);
		weights[weightID] += velocity;// prev[prevLayerID] * gradient / (sqrtf(vel[weightID]) + 0.000001f);;

		float prevActGD = weights[weightID] * gradient;
		atomicAdd(&prevError[prevLayerID], prevActGD);
	}


	__global__ void Clear(float* data, float* error) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);
		
		data[ID] = 0.f;
		error[ID] = 0.f;
	}
}