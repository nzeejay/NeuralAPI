
#include <hip/hip_runtime.h>
﻿//Kernel code:
extern "C"
{
	__device__ int getIndex(int x, int y, int z, int X, int Y) {
		return (x + (y * Y)) + (z * X * Y);
	}

	// Device code
	__global__ void Forward(float* data, float* weights, float* prev) {

		__shared__ float thisNode;

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * (blockDim.x * blockDim.y * blockDim.z) + prevLayerID;

		atomicAdd(&thisNode, prev[prevLayerID] * weights[weightID]);

		__syncthreads();

		//activation function
		if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
			if (thisNode > 1)
				thisNode = 1.f;
			else if (thisNode < 0)
				thisNode = 0.f;

			data[thisLayerID] = thisNode;
		}
	}
}