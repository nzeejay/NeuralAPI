
#include <hip/hip_runtime.h>
﻿//Kernel code:
extern "C"
{
	__device__ int getIndex(int x, int y, int z, int X, int Y) {
		return (x + (y * Y)) + (z * X * Y);
	}

	__device__ float sigmoid(float f) {
		return 1 / (1 + __expf(-f));
	}

	__device__ float sigmoidDer(float f) {
		return f * (1 - f);
	}

	//activations
	__global__ void Sigmoid(float* data, float* bias) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		data[ID] = sigmoid(data[ID] + bias[ID]);
	}

	// Device code
	__global__ void Forward(float* data, float* weights, float* prev) {

		//__shared__ float thisNode;
		//
		//__syncthreads();

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * (blockDim.x * blockDim.y * blockDim.z) + prevLayerID;

		float val = prev[prevLayerID] * weights[weightID];

		//printf("%f \r\n", val);

		atomicAdd(&data[thisLayerID], val);

		//__syncthreads();
		//
		////activation function
		//if (threadIdx.x + threadIdx.y + threadIdx.z == 0) 
		//	data[thisLayerID] = sigmoid(thisNode);

	}

	__global__ void Backprop(float* data, float* weights, float* bias, float* prev, float* error, float* prevError, float* vel, float step, float mu) {
		int blockSize = (blockDim.x * blockDim.y * blockDim.z);

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * blockSize + prevLayerID;

		float prevActGD = weights[weightID] * sigmoidDer(prev[prevLayerID]) * error[thisLayerID];
		atomicAdd(&prevError[prevLayerID], prevActGD);

		float gradient = (step * sigmoidDer(prev[prevLayerID]) * 2 * error[thisLayerID]);// * (1 + powf(error[thisLayerID], 2));

		float velocity = vel[weightID] * 0.001f - (prev[prevLayerID] * gradient); 
		vel[weightID] = velocity;
		weights[weightID] += velocity;
		bias[thisLayerID] += gradient;
		
		//printf("%i %i %i %f %f\r\n", thisLayerID, prevLayerID, weightID, weightGD, prevActGD);
	}

	__global__ void Clear(float* data, float* error) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);
		
		data[ID] = 0.f;
		error[ID] = 0.f;
	}
}